#include "hip/hip_runtime.h"
/*
Kernels for gelu forward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt gelu_forward.cu -o gelu_forward

If encountering "error: identifier "M_PI" is undefined", add the following lines to the top of the file:

#define _USE_MATH_DEFINES
#include <math.h>  OR  #include <cmath>

version 1 is naive CPU port
./gelu_forward 1

version 2 is bfloat16 with the Packed128 data structure
./gelu_forward 2
*/

size_t B = 1024;

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <vector>
#include <algorithm>

//#define ENABLE_BF16
#define ENABLE_FP32
#include "common.h"

constexpr unsigned int latency_threshold_side = 630;
constexpr int num_reads = 16;
constexpr int read_offset = 128;
constexpr bool enable_compression = false;

int num_sms;
int num_threads_per_sm;
int num_near = 0;
int num_far = 0;
unsigned char* sm_side;
unsigned char *block_is_far;
unsigned int* num_blocks_active;

__constant__ unsigned char c_sm_side[256];
__constant__ unsigned char c_is_far[1024];
__constant__ unsigned char c_to_next[1024];

// ----------------------------------------------------------------------------

__global__ void page_side_latency(const unsigned char* data, unsigned int* latencies, unsigned int* smid, size_t num_chunks, size_t granularity) {
    if (threadIdx.x > 0) return;

    unsigned int smid_value;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid_value));
    *smid = smid_value;

    for (int i = 0; i < num_chunks; i++) {
        const unsigned char* address = &data[i*granularity];
        unsigned int clock_start, clock_end;

        #pragma unroll num_reads
        for (int k = 0; k < num_reads; k++) {
            asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_start) :: "memory");
            unsigned char value = *address;
            if (value != 0) return; // dummy exit to avoid dead code optimisation (memset to always be 0)
            asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_end) :: "memory");

            unsigned int latency = (clock_end < clock_start) ? ((0xFFFFFFFF - clock_start) + clock_end) : (clock_end - clock_start);
            latencies[i*num_reads + k] = latency;
            address += read_offset;
        }
    }
}

__global__ void latency_kernel(const unsigned char* data, unsigned char *block_is_far, unsigned int latency_threshold_far, int N) {
    constexpr size_t num_fetches = 8*2048;
    if (threadIdx.x > 0) return;
    int is_far = 0;
    #pragma unroll 1
    for (int i = 0; i < num_fetches; i++) {
        unsigned int clock_start, clock_end;
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_start));
        unsigned char value = data[i*512];
        if (value != 0) return; // dummy exit to avoid dead code optimisation (memset to always be 0)
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_end));
        unsigned int latency = (clock_end < clock_start) ? ((0xFFFFFFFF - clock_start) + clock_end) : (clock_end - clock_start);

        is_far += (latency >= latency_threshold_far) ? 1 : 0;
        if ((i % 8) == 7) {
            block_is_far[i/8] = (is_far > 4) ? 1 : 0;
            is_far = 0;
        }
    }
}

__global__ void sm_kernel(const unsigned char* data, const unsigned char *block_is_far, unsigned char* sm_side, unsigned int latency_threshold_far) {
    if (threadIdx.x > 0) return; // 1st thread of every block only
    unsigned int smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));

    unsigned int latencies[8];
    unsigned int latencies_2[8];

    #pragma unroll 1
    for (int k = 0; k < 8; k++) {
        int i = blockIdx.x * 8 + k;
        unsigned int clock_start, clock_end;
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_start));
        unsigned char value = data[i*512];
        if (value != 0) return; // dummy exit to avoid dead code optimisation (memset to always be 0)
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_end));
        unsigned int latency = (clock_end < clock_start) ? ((0xFFFFFFFF - clock_start) + clock_end) : (clock_end - clock_start);
        latencies[k] = latency;
        __nanosleep(10*smid);
    }

    #pragma unroll 1
    for (int k = 0; k < 8; k++) {
        int i = blockIdx.x * 8 + k + 512*8;
        unsigned int clock_start, clock_end;
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_start));
        unsigned char value = data[i*512];
        if (value != 0) return; // dummy exit to avoid dead code optimisation (memset to always be 0)
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock_end));
        unsigned int latency = (clock_end < clock_start) ? ((0xFFFFFFFF - clock_start) + clock_end) : (clock_end - clock_start);
        latencies_2[k] = latency;
        __nanosleep(10*smid);
    }

    // Find the median of latencies array (in CUDA manually)
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            if (latencies[j] > latencies[j+1]) {
                unsigned int temp = latencies[j];
                latencies[j] = latencies[j+1];
                latencies[j+1] = temp;
            }
            if (latencies_2[j] > latencies_2[j+1]) {
                unsigned int temp = latencies_2[j];
                latencies_2[j] = latencies_2[j+1];
                latencies_2[j+1] = temp;
            }
        }
    }
    unsigned int median_latency = latencies[3];
    unsigned int median_latency_2 = latencies_2[3];
    //printf("SM %u: %u/%u\n", smid, median_latency, median_latency_2);

    sm_side[smid] = (median_latency > median_latency_2) ? block_is_far[blockIdx.x] : !block_is_far[blockIdx.x];
}

void clear_l2() {
    // Get actual L2 size via CUDA
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 4;
        cudaCheck(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size);
}

// ----------------------------------------------------------------------------
// CUDA memory allocation with compressible memory support

hipMemAllocationProp get_allocation_constraints(size_t &granularity, bool &use_compression)
{
    int compression_available;
    hipDeviceGetAttribute(&compression_available, CU_DEVICE_ATTRIBUTE_GENERIC_COMPRESSION_SUPPORTED, 0);
    use_compression = use_compression && compression_available;

    hipMemAllocationProp prop = {};
    memset(&prop, 0, sizeof(hipMemAllocationProp));
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = 0; // force device 0 for now
    prop.allocFlags.compressionType = use_compression ? CU_MEM_ALLOCATION_COMP_GENERIC : 0;

    assert(!hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum));
    assert(granularity == 2048*1024); // todo - currently only support 2MiB granularity
    return prop;
}

void allocateCompressible(void **addr, size_t size, bool use_compression)
{
    //cudaCheck(hipMalloc(&addr, size); return;

    // todo - hack required to avoid initialisation issues (?)
    cudaCheck(hipSetDevice(0));

    size_t granularity;
    hipMemAllocationProp prop = get_allocation_constraints(granularity, use_compression);
    size = ((size - 1) / granularity + 1) * granularity;
    size_t num_chunks = size / granularity;

    hipDeviceptr_t dptr;
    assert(!hipMemAddressReserve(&dptr, size, 0, 0, 0));
    // todo - make sure the virtual address is 4MiB aligned (2x granularity)
    // so that different allocations can match by starting with the same hash

    // allocate each chunk/page of minimum granularity (must be 2MiB for now)
    std::vector <hipMemGenericAllocationHandle_t> allocationHandles(num_chunks);
    for (size_t i = 0; i < num_chunks; i++) {
        assert(!hipMemCreate(&allocationHandles[i], granularity, &prop, 0));
        if (use_compression) {
            hipMemAllocationProp allocationProp = {};
            assert(!hipMemGetAllocationPropertiesFromHandle(&allocationProp, allocationHandles[i]));
            assert(allocationProp.allocFlags.compressionType == CU_MEM_ALLOCATION_COMP_GENERIC);
        }
    }
    // initial mapping which we'll use to determine the hashing of each page/chunk before remapping
    for (size_t i = 0; i < num_chunks; i++) {
        assert(!hipMemMap(dptr + i * granularity, granularity, 0, allocationHandles[i], 0));
    }
    // make allocation readable & writable (on the virtual memory range, not the physical one)
    hipMemAccessDesc accessDescriptor;
    accessDescriptor.location.id = prop.location.id;
    accessDescriptor.location.type = prop.location.type;
    accessDescriptor.flags = hipMemAccessFlagsProtReadWrite;
    assert(!hipMemSetAccess(dptr, size, &accessDescriptor, 1));
    assert(!hipMemsetD8(dptr, 0, size)); // memset allocation to 0
    clear_l2(); // make sure we always miss in the L2 for our allocation

    // alloc latencies array with host cuda malloc
    unsigned int* smid;
    unsigned int* latencies;
    assert(!hipHostAlloc((void**)&smid, sizeof(unsigned int), 0));
    assert(!hipHostAlloc((void**)&latencies, num_chunks * num_reads * sizeof(unsigned int), 0));

    // launch kernel to measure latency
    page_side_latency<<<1, 32>>>((const unsigned char*)dptr, latencies, smid, num_chunks, granularity);
    cudaCheck(hipDeviceSynchronize());

    // print all the latencies
    //printf("num_chunks: %lu (smid: %u)\n", num_chunks, *smid);
    for (int i = 0; i < num_chunks; i++) {
        for (int n = 0; n < num_reads; n++) {
            //printf("%d[%d]: %d\n", i, n, latencies[i*num_reads + n]);
        }
    }

    // get the median latency for each chunk
    // and save them for further use
    for (int i = 0; i < num_chunks; i++) {
        std::vector<unsigned int> chunk_latencies;
        for (int n = 0; n < num_reads; n++) {
            chunk_latencies.push_back(latencies[i*num_reads + n]);
        }
        std::sort(chunk_latencies.begin(), chunk_latencies.end());
        unsigned int median_latency = chunk_latencies[num_reads / 2];
        //printf("Chunk %d median latency: %u\n", i, median_latency);
        // save median latency
        latencies[i] = median_latency;
    }

    // Get the average of all the medians
    unsigned int total_median_latency = 0;
    for (int i = 0; i < num_chunks; i++) {
        total_median_latency += latencies[i];
    }
    unsigned int average_median_latency = total_median_latency / num_chunks;
    printf("Average median latency: %u\n", average_median_latency);

    // unmap our initial unoptimised mapping
    for (size_t i = 0; i < num_chunks; i++) {
        assert(!hipMemUnmap(dptr + i * granularity, granularity));
    }

    // Is bit 21 of the virtual address true? If so, start with far allocations
    // todo - force this to always be true (or false or whatever)
    int start_far = (dptr & (1 << 21)) ? 1 : 0;
    printf("Start far: %d\n", start_far);

    // Remap alternating "far" and "near" allocations
    // i.e. if latencies[i] > average_median_latency, then far, else near
    int current_near = 0;
    int current_far = 0;
    int i = 0;
    while (i < num_chunks) {
        if ((i % 2) == start_far) {
            while (current_far < num_chunks && latencies[current_far] <= average_median_latency) {
                current_far++;
            }
            if (current_far >= num_chunks) break;
            assert(!hipMemMap(dptr + i * granularity, granularity, 0, allocationHandles[current_far], 0));
            current_far++;
        } else {
            while (current_near < num_chunks && latencies[current_near] > average_median_latency) {
                current_near++;
            }
            if (current_near >= num_chunks) break;
            assert(!hipMemMap(dptr + i * granularity, granularity, 0, allocationHandles[current_near], 0));
            current_near++;
        }
        i++;
    }

    int current = min(current_near, current_far);
    while(i < num_chunks) {
        if ((current_near >= num_chunks && latencies[current] > latency_threshold_side)
         || (current_far >= num_chunks && latencies[current] <= latency_threshold_side)) {
            assert(!hipMemMap(dptr + i * granularity, granularity, 0, allocationHandles[current], 0));
            i++;
        }
        current++;
    }

    assert(!hipMemSetAccess(dptr, size, &accessDescriptor, 1));

    cudaCheck(hipHostFree(smid));
    cudaCheck(hipHostFree(latencies));
    printf("post-free\n");

    // Release handles
    for (size_t i = 0; i < num_chunks; i++) {
        assert(!hipMemRelease(allocationHandles[i]));
    }

    printf("post-release\n");
    *addr = (void*)dptr;
}

void freeCompressible(void *ptr, size_t size, bool UseCompressibleMemory)
{
    // hipFree(ptr); return;

    if (ptr == NULL)
        return;
    size_t granularity;
    hipMemAllocationProp prop = get_allocation_constraints(granularity, UseCompressibleMemory);

    size_t chunks = ceil_div(size, granularity);
    for (size_t i = 0; i < chunks; i++) {
        assert(!hipMemUnmap((hipDeviceptr_t)ptr + i * granularity, granularity));
    }
}

// ----------------------------------------------------------------------------
// CPU code reference

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)

void gelu_forward_cpu(float* out, const float* inp, int N) {
    for (int i = 0; i < N; i++) {
        float x = inp[i];
        float cube = 0.044715f * x * x * x;
        out[i] = 0.5f * x * (1.0f + tanhf(GELU_SCALING_FACTOR * (x + cube)));
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// elementwise ops are nice and ez
__global__ void gelu_forward_kernel1(floatX* out, const floatX* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}

// Optimised with option to use optimised HW TANH instruction by default
__global__ void gelu_forward_kernel3(floatX* out, const floatX* inp, int N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (idx >= N) { return; }

    x128 packed_out;
    x128 packed_inp = load128cs(inp + idx); // load and do not keep in cache
    for(int k = 0; k < packed_inp.size; ++k) {
        float xi = (float)packed_inp[k];
        float cube = 0.044715f * xi * xi * xi;

        float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
        #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
        asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
        #else
        tanh_in_out = tanhf(tanh_in_out);
        #endif

        // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
        float half_xi = 0.5f * xi;
        packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
    }
    // store instead of storecs (without cache streaming) in case it is useful for the
    // data to be in the cache for the next operation after this GeLU
    store128(out + idx, packed_out);
}

constexpr int per_clean_iter = 8;
constexpr int parallel_blocks = 3;

// Optimised with option to use optimised HW TANH instruction by default
__global__ __launch_bounds__(256, parallel_blocks)
void gelu_forward_kernel4(floatX* __restrict__ out, const floatX* __restrict__ inp,
                          int iter_per_block, int elements_per_block, int num_near, int num_far,
                          int start_chunk_extra, int iter_per_block_extra, int elements_per_block_extra,
                          unsigned int* __restrict__ num_blocks_active, int N) {
    // get SM id
    unsigned int smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    unsigned int blocks_per_sm = parallel_blocks; //(2048 / blockDim.x);

    __shared__ unsigned int shared_block_in_sm;
    if (threadIdx.x == 0) {
        shared_block_in_sm  = atomicInc(num_blocks_active + smid, blocks_per_sm-1);
    }

    unsigned int sm_side_raw = c_sm_side[smid];
    unsigned int sm_side_near = sm_side_raw & 1;
    unsigned int sm_side_id = sm_side_raw >> 1;
    unsigned int min_per_side = min(num_near, num_far);

    __syncthreads();
    unsigned int block_in_sm = shared_block_in_sm;
    int effective_block_id = (sm_side_id * blocks_per_sm) + block_in_sm;
    //assert(block_in_sm < parallel_blocks);

    if (sm_side_id >= min_per_side) {
        // todo - stragglers do X% + dynamic???
        // use nanosleep to make sure all the other thread blocks have launched
        // and this won't result in this SM getting more than its expected number of thread blocks
        __nanosleep(20000);
        return;

        effective_block_id -= min_per_side * blocks_per_sm;
        size_t idx = start_chunk_extra * (4096 / sizeof(floatX));
        idx += (effective_block_id * elements_per_block_extra) + (threadIdx.x * x128::size);

        #pragma unroll 4
        for(int j = 0; j < iter_per_block_extra; j++) {
            x128 packed_out;
            x128 packed_inp = load128cs(inp + idx); // load and do not keep in cache
            #pragma unroll
            for(int k = 0; k < packed_inp.size; ++k) {
                float xi = (float)packed_inp[k];
                float cube = 0.044715f * xi * xi * xi;

                float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
                #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
                asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
                #else
                tanh_in_out = tanhf(tanh_in_out);
                #endif

                // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
                float half_xi = 0.5f * xi;
                packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
            }
            // store instead of storecs (without cache streaming) in case it is useful for the
            // data to be in the cache for the next operation after this GeLU
            store128(out + idx, packed_out);
            idx += blockDim.x * x128::size;
        }
        return;
    }

    size_t idx = (effective_block_id * elements_per_block) + (threadIdx.x * x128::size);

    //if (smid != 43 || block_in_sm != 0 || threadIdx.x >= 32) {
    //    return;
    //}

    int j = 0;
    size_t addr = (size_t)(inp + idx);
    int chunk_id = (addr >> 12) & 1023;
    unsigned char is_far = c_is_far[chunk_id];

    //while (is_far != (sm_side_near ^ (effective_block_id & 1))) {
    //while (is_far != sm_side_near) {
    while (is_far == sm_side_near) {
        idx += blockDim.x * x128::size;
        chunk_id = (chunk_id + 1) & 1023;
        is_far = c_is_far[chunk_id];
        j++;
    }

    int clean_iters = (iter_per_block / (2 * per_clean_iter)) - 1;

    #pragma unroll 1
    for (int x = 0; x < clean_iters; x++) {
        x128 packed_inps[per_clean_iter];
        //const floatX* in_addrs[per_clean_iter];
        floatX* out_addrs[per_clean_iter];

        #pragma unroll
        for (int y = 0; y < per_clean_iter; y++) {
            //in_addrs[y] = inp + idx;
            packed_inps[y] = load128cs(inp + idx); // load and do not keep in cache
            out_addrs[y] = out + idx;

            int to_next = c_to_next[chunk_id];
            idx += to_next * blockDim.x * x128::size;
            chunk_id = (chunk_id + to_next) & 1023;
            j += to_next;
        }

        __syncthreads();
        __threadfence_block();

        #pragma unroll
        for (int y = 0; y < per_clean_iter; y++) {
            //packed_inps[y] = load128cs(in_addrs[y]); // load and do not keep in cache
        }

        #pragma unroll
        for (int y = 0; y < per_clean_iter; y++) {
            x128 packed_out;
            x128 packed_inp = packed_inps[y];
            #pragma unroll
            for(int k = 0; k < packed_inp.size; ++k) {
                float xi = (float)packed_inp[k];
                float cube = 0.044715f * xi * xi * xi;

                float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
                #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
                asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
                #else
                tanh_in_out = tanhf(tanh_in_out);
                #endif

                // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
                float half_xi = 0.5f * xi;
                packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
            }
            // store instead of storecs (without cache streaming) in case it is useful for the
            // data to be in the cache for the next operation after this GeLU
            store128(out_addrs[y], packed_out);
        }
    }

    #pragma unroll 4
    while(j < iter_per_block) {
        /*
        unsigned int clock, clock2;
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock));
        x128 packed_inp = load128cs(inp + idx); // load and do not keep in cache
        if (packed_inp[0] == (floatX)999999.99999f) {
            return; // HACK for timing
        }
        asm volatile("mov.u32 %0, %%clock;" : "=r"(clock2));
        unsigned int elapsed_clocks = clock2 - clock;
        if (threadIdx.x == 0) {
            printf("%d: %d (is_far: %d / sm_side_near: %d)\n", idx, elapsed_clocks, is_far, sm_side_near);
        }
        */


        x128 packed_out;
        x128 packed_inp = load128cs(inp + idx); // load and do not keep in cache
        #pragma unroll
        for(int k = 0; k < packed_inp.size; ++k) {
            float xi = (float)packed_inp[k];
            float cube = 0.044715f * xi * xi * xi;

            float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
            #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
            asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
            #else
            tanh_in_out = tanhf(tanh_in_out);
            #endif

            // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
            float half_xi = 0.5f * xi;
            packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
        }
        // store instead of storecs (without cache streaming) in case it is useful for the
        // data to be in the cache for the next operation after this GeLU
        store128(out + idx, packed_out);

        int to_next = c_to_next[chunk_id];
        idx += to_next * blockDim.x * x128::size;
        chunk_id = (chunk_id + to_next) & 1023;
        j += to_next;

        //idx += blockDim.x * x128::size; // = 1 chunk of 4KiB
        //chunk_id = (chunk_id + 1) & 1023;
    }
}

__global__ __launch_bounds__(256, parallel_blocks)
void gelu_forward_kernel5(floatX* out, const floatX* inp,
                          int clean_iter_per_block, int plausible_iter_per_block,
                          int iter_per_block, int elements_per_block, int num_near, int num_far,
                          int start_chunk_extra, int iter_per_block_extra, int elements_per_block_extra,
                          unsigned int* num_blocks_active, int N) {
    // get SM id
    unsigned int smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    unsigned int blocks_per_sm = parallel_blocks; //(2048 / blockDim.x);

    int elements_per_4KiB = 4*1024/sizeof(floatX);
    int elements_per_2MiB = 2*1024*1024/sizeof(floatX);

    __shared__ unsigned int shared_block_in_sm;
    if (threadIdx.x == 0) {
        shared_block_in_sm  = atomicInc(num_blocks_active + smid, blocks_per_sm-1);
    }

    unsigned int sm_side_raw = c_sm_side[smid];
    unsigned int sm_side_is_near = sm_side_raw & 1;
    unsigned int sm_side_id = sm_side_raw >> 1;
    unsigned int min_per_side = min(num_near, num_far);

    __syncthreads();
    unsigned int block_in_sm = shared_block_in_sm;
    int effective_block_id = (sm_side_id * blocks_per_sm) + block_in_sm;
    //assert(block_in_sm < parallel_blocks);

    if (sm_side_id >= min_per_side) {
        // todo - stragglers do X% + dynamic???
        __nanosleep(20000);
        return;
    }

    int blocks_per_side = min_per_side * blocks_per_sm;
    int stride = elements_per_4KiB * blocks_per_side;
    unsigned int idx = effective_block_id * elements_per_4KiB + threadIdx.x * x128::size;

    floatX* original_out = out;

    int chunk_id = ((size_t)(inp + idx) >> 12) & 1023;
    idx += (chunk_id >= 512) ? elements_per_2MiB : 0;
    chunk_id &= 511;

    #pragma unroll 1
    for (int x = 0; x < clean_iter_per_block; x += per_clean_iter) {
        x128 packed_inps[per_clean_iter];
        //const floatX* in_addrs[per_clean_iter];
        floatX* out_addrs[per_clean_iter];

        #pragma unroll
        for (int y = 0; y < per_clean_iter; y++) {
            unsigned int idx_plus_2MiB = idx + elements_per_2MiB;
            unsigned int idx2 = (c_is_far[chunk_id] == sm_side_is_near) ? idx_plus_2MiB : idx;
            packed_inps[y] = load128cs(inp + idx2);
            out_addrs[y] = out + idx2;

            chunk_id = (chunk_id + blocks_per_side);
            idx += stride + ((chunk_id >= 512) ? elements_per_2MiB : 0);
            chunk_id &= 511;
        }
        inp += idx;
        out += idx;
        idx = 0;

        __syncthreads();
        __threadfence_block();

        #pragma unroll
        for (int y = 0; y < per_clean_iter; y++) {
            x128 packed_out;
            x128 packed_inp = packed_inps[y];
            #pragma unroll
            for(int k = 0; k < packed_inp.size; ++k) {
                float xi = (float)packed_inp[k];
                float cube = 0.044715f * xi * xi * xi;

                float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
                #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
                asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
                #else
                tanh_in_out = tanhf(tanh_in_out);
                #endif

                // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
                float half_xi = 0.5f * xi;
                packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
            }
            // store instead of storecs (without cache streaming) in case it is useful for the
            // data to be in the cache for the next operation after this GeLU
            store128(out_addrs[y], packed_out);
        }
    }

    size_t real_idx = (size_t)(out - original_out);

    for(int j = 0; j < plausible_iter_per_block; j++) {
        unsigned int offset = (c_is_far[chunk_id] == sm_side_is_near) ? elements_per_2MiB : 0;

        if (real_idx + offset < N) {
            x128 packed_out;
            x128 packed_inp = load128cs(inp + offset + idx);
            #pragma unroll
            for(int k = 0; k < packed_inp.size; ++k) {
                float xi = (float)packed_inp[k];
                float cube = 0.044715f * xi * xi * xi;

                float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
                #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
                asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
                #else
                tanh_in_out = tanhf(tanh_in_out);
                #endif

                // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
                float half_xi = 0.5f * xi;
                packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
            }
            // store instead of storecs (without cache streaming) in case it is useful for the
            // data to be in the cache for the next operation after this GeLU
            store128(out + offset + idx, packed_out);
        }

        chunk_id += blocks_per_side;
        idx += stride + (chunk_id >= 512 ? elements_per_2MiB : 0);
        chunk_id &= 511;
    }
}




// ----------------------------------------------------------------------------
// kernel launcher

void gelu_forward1(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size);
    gelu_forward_kernel1<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward3(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size * x128::size);
    gelu_forward_kernel3<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

constexpr float use_wasted_multiplier = 0.0f;

void gelu_forward4(floatX* out, const floatX* inp, int N, int block_size) {
    /*int blocks_per_sm = num_threads_per_sm / block_size;
    int grid_size = num_sms * blocks_per_sm;
    int iter_per_block = ceil_div(N, grid_size * block_size * x128::size);
    int elements_per_block = iter_per_block * block_size * x128::size;*/

    block_size = 256; // hack
    //int blocks_per_sm = (num_threads_per_sm / block_size);
    int blocks_per_sm = parallel_blocks; // hack

    int elements_per_4kib_chunk = 4096 / sizeof(floatX);
    int num_4kib_chunks = ceil_div(N, elements_per_4kib_chunk);
    int min_per_side = min(num_near, num_far);

    int wasted_SMs = num_sms - (2 * min_per_side);
    float percentage_wasted = (float)wasted_SMs / (float)num_sms;
    float adjusted_used = 1.0f - (use_wasted_multiplier * percentage_wasted);

    int num_4kib_chunks_main = (int)ceilf((float)num_4kib_chunks * adjusted_used);
    int num_4kib_chunks_extra = num_4kib_chunks - num_4kib_chunks_main;

    int effective_num_blocks = min_per_side * blocks_per_sm;
    int iter_per_block = ceil_div(num_4kib_chunks_main, effective_num_blocks);
    int elements_per_block = elements_per_4kib_chunk * iter_per_block;

    int effective_num_blocks_extra = wasted_SMs * blocks_per_sm;
    int iter_per_block_extra = ceil_div(num_4kib_chunks_extra, effective_num_blocks_extra);
    int elements_per_block_extra = elements_per_4kib_chunk * iter_per_block_extra;

    /*
    printf("\n==========\n");
    printf("N: %d\n", N);
    printf("num_4kib_chunks: %d\n", num_4kib_chunks);
    printf("num_4kib_chunks_main: %d\n", num_4kib_chunks_main);
    printf("num_4kib_chunks_extra: %d\n", num_4kib_chunks_extra);
    printf("iter_per_block: %d\n", iter_per_block);
    printf("elements_per_block: %d\n", elements_per_block);
    printf("iter_per_block_extra: %d\n", iter_per_block_extra);
    printf("elements_per_block_extra: %d\n", elements_per_block_extra);
    printf("effective_num_blocks: %d\n", effective_num_blocks);
    printf("effective_num_blocks_extra: %d\n", effective_num_blocks_extra);
    printf("SMs: %d, min_per_side: %d, wasted_SMs: %d\n", num_sms, min_per_side, wasted_SMs);
    printf("percentage_wasted: %f\n", percentage_wasted);
    printf("adjusted_used: %f\n", adjusted_used);
    printf("==========\n\n");
    */

    int inner_loop_iter = elements_per_4kib_chunk / (block_size * x128::size);
    assert(inner_loop_iter == 1);

    //hipMemset(num_blocks_active, 0, num_sms * 64 * sizeof(unsigned int));

    int grid_size = num_sms * blocks_per_sm;
    gelu_forward_kernel4<<<grid_size, block_size>>>(out, inp, iter_per_block, elements_per_block, num_near, num_far,
                                                    num_4kib_chunks_main, iter_per_block_extra, elements_per_block_extra,
                                                    num_blocks_active, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward5(floatX* out, const floatX* inp, int N, int block_size) {
    block_size = 256; // hack
    int blocks_per_sm = parallel_blocks; // hack

    int elements_per_4kib_chunk = 4096 / sizeof(floatX);
    int num_4kib_chunks = ceil_div(N, elements_per_4kib_chunk);
    int min_per_side = min(num_near, num_far);

    int wasted_SMs = num_sms - (2 * min_per_side);
    float percentage_wasted = (float)wasted_SMs / (float)num_sms;
    float adjusted_used = 1.0f - (use_wasted_multiplier * percentage_wasted);
    int num_4kib_chunks_main = (int)ceilf((float)num_4kib_chunks * adjusted_used);


    int effective_num_blocks = 2 * min_per_side * blocks_per_sm;
    int iter_per_block = ceil_div(num_4kib_chunks_main, effective_num_blocks);
    int elements_per_block = elements_per_4kib_chunk * iter_per_block;

    int safe_4kib_chunks_main = max(0, num_4kib_chunks_main - 512);
    int clean_iter_per_block = ((safe_4kib_chunks_main / effective_num_blocks) / per_clean_iter) * per_clean_iter;

    int plausible_4kib_chunks_main = ceil_div(num_4kib_chunks_main, 1024) * 1024; // todo - optimise
    int plausible_iter_per_block = (plausible_4kib_chunks_main / effective_num_blocks) - clean_iter_per_block + 1;


    int num_4kib_chunks_extra = num_4kib_chunks - num_4kib_chunks_main;
    int effective_num_blocks_extra = wasted_SMs * blocks_per_sm;
    int iter_per_block_extra = wasted_SMs ? ceil_div(num_4kib_chunks_extra, effective_num_blocks_extra) : 0;
    int elements_per_block_extra = elements_per_4kib_chunk * iter_per_block_extra;

    int inner_loop_iter = elements_per_4kib_chunk / (block_size * x128::size);
    assert(inner_loop_iter == 1);

    //hipMemset(num_blocks_active, 0, num_sms * 64 * sizeof(unsigned int));
    int grid_size = num_sms * blocks_per_sm;
    gelu_forward_kernel5<<<grid_size, block_size>>>(out, inp, clean_iter_per_block, plausible_iter_per_block,
                                                    iter_per_block, elements_per_block, num_near, num_far,
                                                    num_4kib_chunks_main, iter_per_block_extra, elements_per_block_extra,
                                                    num_blocks_active, N);
    cudaCheck(hipGetLastError());
}


// kernel version dispatch
void gelu_forward(int kernel_num,
                  floatX* out,
                  const floatX* inp,
                  int B, int T, int C,
                  int block_size) {
    switch (kernel_num) {
        case 1:
            gelu_forward1(out, inp, B * T * C, block_size);
            break;
        case 3:
            gelu_forward3(out, inp, B * T * C, block_size);
            break;
        case 4:
            gelu_forward4(out, inp, B * T * C, block_size);
            break;
        case 5:
            gelu_forward5(out, inp, B * T * C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, const char **argv) {
    setup_main();

    size_t T = 1024;
    size_t C = 768;

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* inp = make_random_float(B * T * C);

    // read kernel_num from command line
    int kernel_num = (argc > 1) ? atoi(argv[1]) : 1;
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    gelu_forward_cpu(out, inp, B * T * C);




    cudaCheck(hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0));
    cudaCheck(hipDeviceGetAttribute(&num_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
    printf("Number of SMs: %d\n", num_sms);

    assert(!hipHostAlloc((void**)&block_is_far, 2048 * sizeof(unsigned char), 0));
    cudaCheck(hipHostMalloc((void**)&sm_side, num_sms * sizeof(unsigned char)));
    cudaCheck(hipHostMalloc((void**)&num_blocks_active, num_sms * 64 * sizeof(unsigned int)));
    hipMemset(num_blocks_active, 0, num_sms * 64 * sizeof(unsigned int));






    // move to GPU
    floatX *d_out, *d_inp;
    allocateCompressible((void**)&d_out, B * T * C * sizeof(floatX) * 2, enable_compression);
    allocateCompressible((void**)&d_inp, B * T * C * sizeof(floatX) * 2, enable_compression);

    clear_l2();
    latency_kernel<<<1, 32>>>((unsigned char*)d_out, block_is_far, 690 /* todo: dynamic */, B * T * C);
    cudaCheck(hipDeviceSynchronize());

    for (int i = 0; i < 2048; i++) {
        printf("%d", block_is_far[i]);
        if ((i % 512) == 511) {
            printf("\n");
        }
    }


    // Let's figure out which SMs are on the "near" and "far" sides
    clear_l2();
    sm_kernel<<<num_sms, 512>>>((unsigned char*)d_out, block_is_far, sm_side, 690);
    cudaCheck(hipDeviceSynchronize());

    unsigned char to_next[1024] = {0};
    for (int i = 0; i < 1024; i++) {
        unsigned char is_far = block_is_far[i];
        for (int j = i+1; j < 2048; j++) {
            if (block_is_far[j] == is_far) {
                to_next[i] = j - i;
                break;
            }
        }
    }

    for (int i = 0; i < num_sms; i++) {
        if (sm_side[i] == 0) {
            sm_side[i] |= num_near << 1;
            num_near++;
        } else {
            sm_side[i] |= num_far << 1;
            num_far++;
        }
        //printf("SM %d: %s/%d\n", i, (sm_side[i] & 1) ? "far" : "near", sm_side[i] >> 1);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_sm_side), sm_side, num_sms * sizeof(unsigned char));
    hipMemcpyToSymbol(HIP_SYMBOL(c_is_far), block_is_far, 1024 * sizeof(unsigned char));
    hipMemcpyToSymbol(HIP_SYMBOL(c_to_next), to_next, 1024 * sizeof(unsigned char));
    cudaCheck(memcpy_convert(d_inp, inp, B * T * C));
    cudaCheck(hipDeviceSynchronize());

    // time & validate the kernel at different block sizes
    int block_sizes[] = {256};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        // memset d_out to make sure any correct result isn't from the previous kernel
        hipMemset(d_out, 0, B * T * C * sizeof(floatX));

        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        gelu_forward(kernel_num, d_out, d_inp, B, T, C, block_size);
#if !defined(ENABLE_BF16) && !defined(ENABLE_FP16)
        float tol = 1e-5f;
#else
        float tol = 1e-3f;
#endif
        validate_result(d_out, out, "out", B * T * C, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, gelu_forward,
                                              kernel_num, d_out, d_inp,
                                              B, T, C, block_size);

        // for each (B,T,C) output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = B * T * C * 2 * (int)sizeof(floatX);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;
        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    freeCompressible(d_out, B * T * C * sizeof(floatX), enable_compression);
    freeCompressible(d_inp, B * T * C * sizeof(floatX), enable_compression);
    free(out);
    free(inp);
    return 0;
}